
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;
void __global__ add(const double *x, const double *y, double *z);
void check(const double *z, const int N);

int main(void) {
  // 原书中的N值为1亿（作者所用显卡的显存有8G），已经超过我现用显卡的显存（2G）；
  // 再加上其他方面对显存的消耗，所以这里将N值设为7千万。
  const int N = 50000000;
  const int M = sizeof(double) * N;
  // 下面是在主机中分配内存
  double *h_x = (double*)malloc(M);
  if (h_x == NULL) {
    printf("h_x: Failed to allocate host memory!\n");
    return 1;
  }
  double *h_y = (double*)malloc(M);
  if (h_y == NULL) {
    printf("h_y: Failed to allocate host memory!\n");
    free(h_x);
    return 1;
  }
  double *h_z = (double*)malloc(M);
  if (h_z == NULL) {
    printf("h_z: Failed to allocate host memory!\n");
    free(h_x);
    free(h_z);
    return 1;
  }
  
  // 数组元素初始化
  for (int n = 0; n < N; ++n) {
    h_x[n] = a;
    h_y[n] = b;
  }
  
  double *d_x, *d_y, *d_z;
  // 在显卡中分配显存
  hipError_t err;// 判断显存分配是否成功
  err = hipMalloc((void **)&d_x, M);
  if (err != hipSuccess) {
    printf("d_x: hipMemcpy Error!\n");
    free(h_x);
    free(h_y);
    free(h_z);
    return 1;
  }
  err = hipMalloc((void **)&d_y, M);
  if (err != hipSuccess) {
    printf("d_y: hipMemcpy Error!\n");
    free(h_x);
    free(h_y);
    free(h_z);
    hipFree(d_x);
    return 1;
  }
  err = hipMalloc((void **)&d_z, M);
  if (err != hipSuccess) {
    printf("d_z: hipMemcpy Error!\n");
    free(h_x);
    free(h_y);
    free(h_z);
    hipFree(d_x);
    hipFree(d_y);
    return 1;
  }
  // 把主机中的数据复制到显存中
  hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);
  
  // 设置线程块大小为128，网格大小为N/128=546875。
  const int block_size = 128;
  const int grid_size = N / block_size;
  add<<<grid_size, block_size>>>(d_x, d_y, d_z);
  
  // 把计算结果从显存复制到主机
  hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
  // 检查结果
  check(h_z, N);
  
  // 释放内存、显存
  free(h_x);
  free(h_y);
  free(h_z);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  return 0;
}

// 相加
void __global__ add(const double *x, const double *y, double *z) {
  // blockDim.x 的数值等于执行配置中变量 block_size 的数值，在本例中为 128；
  // blockIdx.x 指定一个线程在一个网格中的线程块指标，其取值范围从0到 gridDim.x-1。本例中，即为0到546874；
  // threadIdx.x 指定一个线程在一个线程块中的线程指标，其取值范围从0到 blockDim.x-1。本例中，即为0到127。
  // 显存中的数组与网格、线程块对应，通过下面计算索引值，让每个线程对应的数组进行计算，实现高效的并行计算。
  const int n = blockDim.x * blockIdx.x + threadIdx.x;
  z[n] = x[n] + y[n];
}

// 判断两个浮点数是否相等。注意，不能使用运算符==，而要将两个数的差的绝对值与一个很小的数进行比较。
// 本例中，假定当两个双精度浮点数的差的绝对值小于1e-15（EPISILON）时它们就是相等的。
void check(const double *z, const int N) {
  bool has_error = false;
  for (int n = 0; n < N; ++n) {
    if (fabs(z[n] - c) > EPSILON) {
      has_error = true;
      printf("z[%d] is: %f, c is: %f\n", n, z[n], c);
    }
  }
  printf("%s\n", has_error ? "Has errors": "No errors");
}
